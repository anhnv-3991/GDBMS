#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include "gexpression.h"
#include <string>
#include <sstream>

namespace voltdb {

GExpression::GExpression(ExpressionNode *expression) {
	int size = 0;

	size =	getExpressionLength(expression);

	size_ = size;

	if (size_ > 0) {
		checkCudaErrors(hipMalloc(&expression_, tree_size * sizeof(GTreeNode)));
		createExpression(expression);
	} else {
		size_ = 0;
		expression_ = NULL;
	}
}

bool GExpression::createExpression(ExpressionNode *expression) {
	GTreeNode *tmp_expression = (GTreeNode*)malloc(sizeof(GTreeNode) * size_);

	int root = 0;

	if (!buildPostExpression(tmp_expression, expression, &root))
		return false;

	checkCudaErrors(hipMemcpy(expression_, tmp_expression, sizeof(GTreeNode) * size_, hipMemcpyHostToDevice));
	std::free(tmp_expression);

	return true;
}

void GExpression::free() {
	if (size_ > 0) {
		checkCudaErrors(hipFree(expression_));
		expression_ = NULL;
		size_ = 0;
	}
}

int GExpression::getExpressionLength(ExpressionNode *expression) {
	if (expression == NULL) {
		return 0;
	}

	int left, right;

	left = getExpressionLength(expression->left);
	right = getExpressionLength(expression->right);

	return (1 + left + right);
}

bool GExpression::buildPostExpression(GTreeNode *output_expression, ExpressionNode *expression, int *index) {
	if (expression == NULL)
		return true;

	if (size_ <= *index)
		return false;

	if (!buildPostExpression(output_expression, expression->left, index))
		return false;

	if (!buildPostExpression(output_expression, expression->right, index))
		return false;

	output_expression[*index] = expression->node;
	(*index)++;

	return true;
}

std::string GExpression::debug() const
{
	if (size_ == 0) {
		std::string retval("Empty expression");
		return retval;
	}

	std::ostringstream output;

	GTreeNode *expression_host = (GTreeNode*)malloc(sizeof(GTreeNode) * size_);

	checkCudaErrors(hipMemcpy(expression_host, expression_, sizeof(GTreeNode) * size_, hipMemcpyDeviceToHost));

	for (int i = 0; i < size_; i++)
		output << printNode(expression_host[i], i);

	std::free(expression_host);

	output << "End of expression" << std::endl;

	std::string retval(output.str());

	return retval;
}


std::string GExpression::printNode(GTreeNode node, int index) const
{
	std::ostringstream output;

	output << "[" << index << "]: ";
	switch (node.type) {
	case EXPRESSION_TYPE_OPERATOR_PLUS: {
		output << "Operator PLUS";
		break;
	}
	case EXPRESSION_TYPE_OPERATOR_MINUS: {
		output << "Operator MINUS";
		break;
	}
	case EXPRESSION_TYPE_OPERATOR_MULTIPLY: {
		output << "Operator MULTIPLY";
		break;
	}
	case EXPRESSION_TYPE_OPERATOR_DIVIDE: {
		output << "Operator DIVIDE";
		break;
	}
	case EXPRESSION_TYPE_OPERATOR_NOT: {
		output << "Operator NOT";
		break;
	}
	case EXPRESSION_TYPE_OPERATOR_CONCAT:
	case EXPRESSION_TYPE_OPERATOR_MOD:
	case EXPRESSION_TYPE_OPERATOR_CAST:
	case EXPRESSION_TYPE_OPERATOR_IS_NULL:
	case EXPRESSION_TYPE_COMPARE_LIKE:
	case EXPRESSION_TYPE_COMPARE_IN: {
		output << "Operator unsupported";
		break;
	}
	case EXPRESSION_TYPE_COMPARE_EQUAL: {
		output << "Compare EQUAL";
		break;
	}
	case EXPRESSION_TYPE_COMPARE_NOTEQUAL: {
		output << "Compare NOTEQUAL";
		break;
	}
	case EXPRESSION_TYPE_COMPARE_LESSTHAN: {
		output << "Compare LESSTHAN";
		break;
	}
	case EXPRESSION_TYPE_COMPARE_GREATERTHAN: {
		output << "Compare GREATERTHAN";
		break;
	}
	case EXPRESSION_TYPE_COMPARE_LESSTHANOREQUALTO: {
		output << "Compare LESSTHANOREQUALTO";
		break;
	}
	case EXPRESSION_TYPE_COMPARE_GREATERTHANOREQUALTO: {
		output << "Compare GREATERTHANOREQUALTO";
		break;
	}
	case EXPRESSION_TYPE_CONJUNCTION_AND: {
		output << "Conjunction AND";
		break;
	}
	case EXPRESSION_TYPE_CONJUNCTION_OR: {
		output << "Conjunction OR";
		break;
	}
	case EXPRESSION_TYPE_VALUE_CONSTANT: {
		output << "Value CONSTANT";
		break;
	}
	case EXPRESSION_TYPE_VALUE_PARAMETER: {
		output << "Value PARAMETER";
		break;
	}
	case EXPRESSION_TYPE_VALUE_TUPLE: {
		output << "Value TUPLE : ";
		output << "Column: " << node.column_idx << ":";
		output << "Table: " << node.tuple_idx;
		break;
	}
	case EXPRESSION_TYPE_VALUE_TUPLE_ADDRESS: {
		output << "Value TUPLE ADDRESS";
		break;
	}
	case EXPRESSION_TYPE_VALUE_NULL: {
		output << "Value NULL";
		break;
	}
	case EXPRESSION_TYPE_VALUE_VECTOR: {
		output << "Value VECTOR";
		break;
	}
	case EXPRESSION_TYPE_AGGREGATE_COUNT: {
		output << "Aggregate COUNT";
		break;
	}
	case EXPRESSION_TYPE_AGGREGATE_COUNT_STAR: {
		output << "Aggregate COUNT STAR";
		break;
	}
	case EXPRESSION_TYPE_AGGREGATE_SUM: {
		output << "Aggregate SUM";
		break;
	}
	case EXPRESSION_TYPE_AGGREGATE_MIN: {
		output << "Aggregate MIN";
		break;
	}
	case EXPRESSION_TYPE_AGGREGATE_MAX: {
		output << "Aggregate MAX";
		break;
	}
	case EXPRESSION_TYPE_AGGREGATE_AVG: {
		output << "Aggregate AVG";
		break;
	}
	case EXPRESSION_TYPE_FUNCTION: {
		output << "FUNCTION";
		break;
	}
	case EXPRESSION_TYPE_HASH_RANGE: {
		output << "HASH RANGE";
		break;
	}
	case EXPRESSION_TYPE_OPERATOR_CASE_WHEN: {
		output << "Operator CASE WHEN";
		break;
	}
	case EXPRESSION_TYPE_OPERATOR_ALTERNATIVE: {
		output << "Operator ALTERNATIVE";
		break;
	}
	case EXPRESSION_TYPE_INVALID:
	default: {
		output << "Invalid node";
		break;
	}
	}
	output << std::endl;

	std::string retval(output.str());

	return retval;
}

GExpressionVector::GExpressionVector(std::vector<ExpressionNode*> expression_list)
{
	if (expression_list.size() > 0) {
		exp_num_ = expression_list.size();

		int *exp_size_host = (int*)malloc(sizeof(int) * (exp_num_ + 1));
		int old_size = 0;

		for (int i = 0; i < exp_num_; i++) {
			exp_size_host[i] = old_size;
			old_size += GExpression::getExpressionLength(expression_list[i]);
		}

		exp_size_host[exp_num_] = old_size;

		checkCudaErrors(hipMalloc(&exp_size_, sizeof(int) * (exp_num_ + 1)));
		checkCudaErrors(hipMemcpy(exp_size_, exp_size_host, sizeof(int) * (exp_num_ + 1), hipMemcpyHostToDevice));
		checkCudaErrors(hipMalloc(&expression_, sizeof(GTreeNode) * old_size));

		for (int i = 0; i < exp_num_; i++) {
			GExpression exp(expression_ + exp_size_host[i], exp_size_host[i + 1] - exp_size_host[i]);

			exp.createExpression(expression_list[i]);
		}

		std::free(exp_size_host);
	} else {
		expression_ = NULL;
		exp_size_ = NULL;
		exp_num_ = 0;
	}
}

void GExpressionVector::free()
{
	if (expression_ != NULL) {
		checkCudaErrors(hipFree(expression_));
		expression_ = NULL;
	}

	if (exp_size_ != NULL) {
		checkCudaErrors(hipFree(exp_size_));
		exp_size_ = NULL;
	}

	exp_num_ = 0;
}

std::string GExpressionVector::debug() const
{
	std::ostringstream output;

	output << "Expression list:" << std::endl;
	for (int i = 0; i < exp_num_; i++) {
		GExpression tmp = this->at(i);
		output << "Expression[" << i << "]:" << std::endl << tmp.debug();
	}

	output << "End of the expression list" << std::endl;

	std::string retval(output.str());

	return retval;
}

}
